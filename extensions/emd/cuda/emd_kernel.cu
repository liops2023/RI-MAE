#include "hip/hip_runtime.h"
/**********************************
 * EMD CUDA Kernels (CUDA ≥ 12, PyTorch ≥ 1.11)
 * Original Author : Haoqiang Fan
 * Modifications   : Kaichun Mo → CUDA 12.x port by ChatGPT (2025-04-23)
 *********************************/

#ifndef _EMD_KERNEL
#define _EMD_KERNEL

#include <cmath>
#include <vector>

#include <ATen/ATen.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>   // at::cuda::getApplyGrid
// ──────────────────────────────────────────────────────────────
//  THC headers were removed in PyTorch ≥ 1.11. Use c10/cuda.*
// ──────────────────────────────────────────────────────────────
#include <c10/cuda/CUDAException.h>
#include <c10/cuda/CUDAGuard.h>

// -----------------------------------------------------------------------------
// Helper-macro replacements for deprecated THC counterparts
// -----------------------------------------------------------------------------
#ifndef C10_CUDA_CHECK
#define C10_CUDA_CHECK(err) TORCH_CHECK((err) == hipSuccess, hipGetErrorString(err))
#endif

#define CHECK_CUDA(x)        TORCH_CHECK((x).is_cuda(),       #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x)  TORCH_CHECK((x).is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x)       CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

// gLog style → Torch macros
#define TORCH_CHECK_EQ(a, b) TORCH_CHECK((a) == (b), "Equality check failed: " #a " vs " #b)

/********************************
* Forward kernel for approxmatch
*********************************/

template <typename scalar_t>
__global__ void approxmatch(int b, int n, int m,
                            const scalar_t* __restrict__ xyz1,
                            const scalar_t* __restrict__ xyz2,
                            scalar_t* __restrict__ match,
                            scalar_t* __restrict__ temp) {
    scalar_t* remainL = temp + blockIdx.x * (n + m) * 2;
    scalar_t* remainR = temp + blockIdx.x * (n + m) * 2 + n;
    scalar_t* ratioL  = temp + blockIdx.x * (n + m) * 2 + n + m;
    scalar_t* ratioR  = temp + blockIdx.x * (n + m) * 2 + n + m + n;

    scalar_t multiL, multiR;
    if (n >= m) {
        multiL = 1;
        multiR = static_cast<scalar_t>(n) / static_cast<scalar_t>(m);
    } else {
        multiL = static_cast<scalar_t>(m) / static_cast<scalar_t>(n);
        multiR = 1;
    }

    const int Block = 1024;
    __shared__ scalar_t buf[Block * 4];

    for (int i = blockIdx.x; i < b; i += gridDim.x) {
        for (int j = threadIdx.x; j < n * m; j += blockDim.x) match[i * n * m + j] = 0;
        for (int j = threadIdx.x; j < n;     j += blockDim.x) remainL[j] = multiL;
        for (int j = threadIdx.x; j < m;     j += blockDim.x) remainR[j] = multiR;
        __syncthreads();

        for (int j = 7; j >= -2; --j) {
            scalar_t level = -powf(4.0f, static_cast<float>(j));
            if (j == -2) level = 0;

            // ---- left to right pass -------------------------------------------------
            for (int k0 = 0; k0 < n; k0 += blockDim.x) {
                int k = k0 + threadIdx.x;
                scalar_t x1 = 0, y1 = 0, z1 = 0;
                if (k < n) {
                    x1 = xyz1[i * n * 3 + k * 3 + 0];
                    y1 = xyz1[i * n * 3 + k * 3 + 1];
                    z1 = xyz1[i * n * 3 + k * 3 + 2];
                }
                scalar_t suml = 1e-9f;
                for (int l0 = 0; l0 < m; l0 += Block) {
                    int lend = min(m, l0 + Block) - l0;
                    for (int l = threadIdx.x; l < lend; l += blockDim.x) {
                        scalar_t x2 = xyz2[i * m * 3 + (l0 + l) * 3 + 0];
                        scalar_t y2 = xyz2[i * m * 3 + (l0 + l) * 3 + 1];
                        scalar_t z2 = xyz2[i * m * 3 + (l0 + l) * 3 + 2];
                        buf[l * 4 + 0] = x2;
                        buf[l * 4 + 1] = y2;
                        buf[l * 4 + 2] = z2;
                        buf[l * 4 + 3] = remainR[l0 + l];
                    }
                    __syncthreads();
                    for (int l = 0; l < lend; ++l) {
                        scalar_t x2 = buf[l * 4 + 0];
                        scalar_t y2 = buf[l * 4 + 1];
                        scalar_t z2 = buf[l * 4 + 2];
                        scalar_t d  = level * ((x2 - x1) * (x2 - x1) + (y2 - y1) * (y2 - y1) + (z2 - z1) * (z2 - z1));
                        scalar_t w  = __expf(d) * buf[l * 4 + 3];
                        suml += w;
                    }
                    __syncthreads();
                }
                if (k < n) ratioL[k] = remainL[k] / suml;
            }
            __syncthreads();

            // ---- right to left pass -------------------------------------------------
            for (int l0 = 0; l0 < m; l0 += blockDim.x) {
                int l = l0 + threadIdx.x;
                scalar_t x2 = 0, y2 = 0, z2 = 0;
                if (l < m) {
                    x2 = xyz2[i * m * 3 + l * 3 + 0];
                    y2 = xyz2[i * m * 3 + l * 3 + 1];
                    z2 = xyz2[i * m * 3 + l * 3 + 2];
                }
                scalar_t sumr = 0;
                for (int k0 = 0; k0 < n; k0 += Block) {
                    int kend = min(n, k0 + Block) - k0;
                    for (int k = threadIdx.x; k < kend; k += blockDim.x) {
                        buf[k * 4 + 0] = xyz1[i * n * 3 + (k0 + k) * 3 + 0];
                        buf[k * 4 + 1] = xyz1[i * n * 3 + (k0 + k) * 3 + 1];
                        buf[k * 4 + 2] = xyz1[i * n * 3 + (k0 + k) * 3 + 2];
                        buf[k * 4 + 3] = ratioL[k0 + k];
                    }
                    __syncthreads();
                    for (int k = 0; k < kend; ++k) {
                        scalar_t x1 = buf[k * 4 + 0];
                        scalar_t y1 = buf[k * 4 + 1];
                        scalar_t z1 = buf[k * 4 + 2];
                        scalar_t w  = __expf(level * ((x2 - x1) * (x2 - x1) + (y2 - y1) * (y2 - y1) + (z2 - z1) * (z2 - z1))) * buf[k * 4 + 3];
                        sumr += w;
                    }
                    __syncthreads();
                }
                if (l < m) {
                    sumr *= remainR[l];
                    scalar_t consumption = fminf(remainR[l] / (sumr + 1e-9f), 1.0f);
                    ratioR[l]  = consumption * remainR[l];
                    remainR[l] = fmaxf(0.0f, remainR[l] - sumr);
                }
            }
            __syncthreads();

            // ---- propagate match weights -------------------------------------------
            for (int k0 = 0; k0 < n; k0 += blockDim.x) {
                int k = k0 + threadIdx.x;
                scalar_t x1 = 0, y1 = 0, z1 = 0;
                if (k < n) {
                    x1 = xyz1[i * n * 3 + k * 3 + 0];
                    y1 = xyz1[i * n * 3 + k * 3 + 1];
                    z1 = xyz1[i * n * 3 + k * 3 + 2];
                }
                scalar_t suml = 0;
                for (int l0 = 0; l0 < m; l0 += Block) {
                    int lend = min(m, l0 + Block) - l0;
                    for (int l = threadIdx.x; l < lend; l += blockDim.x) {
                        buf[l * 4 + 0] = xyz2[i * m * 3 + (l0 + l) * 3 + 0];
                        buf[l * 4 + 1] = xyz2[i * m * 3 + (l0 + l) * 3 + 1];
                        buf[l * 4 + 2] = xyz2[i * m * 3 + (l0 + l) * 3 + 2];
                        buf[l * 4 + 3] = ratioR[l0 + l];
                    }
                    __syncthreads();
                    scalar_t rl = ratioL[k];
                    if (k < n) {
                        for (int l = 0; l < lend; ++l) {
                            scalar_t x2 = buf[l * 4 + 0];
                            scalar_t y2 = buf[l * 4 + 1];
                            scalar_t z2 = buf[l * 4 + 2];
                            scalar_t w  = __expf(level * ((x2 - x1) * (x2 - x1) + (y2 - y1) * (y2 - y1) + (z2 - z1) * (z2 - z1))) * rl * buf[l * 4 + 3];
                            match[i * n * m + (l0 + l) * n + k] += w;
                            suml += w;
                        }
                    }
                    __syncthreads();
                }
                if (k < n) remainL[k] = fmaxf(0.0f, remainL[k] - suml);
            }
            __syncthreads();
        }
    }
}

/* -------------------------------------------------------------------------- */
/* ApproxMatch forward host interface                                         */
/* -------------------------------------------------------------------------- */

at::Tensor ApproxMatchForward(const at::Tensor xyz1, const at::Tensor xyz2) {
    const auto b = xyz1.size(0);
    const auto n = xyz1.size(1);
    const auto m = xyz2.size(1);

    TORCH_CHECK_EQ(xyz2.size(0), b);
    TORCH_CHECK_EQ(xyz1.size(2), 3);
    TORCH_CHECK_EQ(xyz2.size(2), 3);
    CHECK_INPUT(xyz1);
    CHECK_INPUT(xyz2);

    auto match = at::zeros({b, m, n}, xyz1.options());
    auto temp  = at::zeros({b, (n + m) * 2}, xyz1.options());

    AT_DISPATCH_FLOATING_TYPES(xyz1.scalar_type(), "ApproxMatchForward", ([&] {
        approxmatch<scalar_t><<<32, 512>>>(
            b, n, m,
            xyz1.data_ptr<scalar_t>(),
            xyz2.data_ptr<scalar_t>(),
            match.data_ptr<scalar_t>(),
            temp.data_ptr<scalar_t>());
    }));
    C10_CUDA_CHECK(hipGetLastError());

    return match;
}

/********************************
* Forward kernel for matchcost
*********************************/

template <typename scalar_t>
__global__ void matchcost(int b, int n, int m,
                          const scalar_t* __restrict__ xyz1,
                          const scalar_t* __restrict__ xyz2,
                          const scalar_t* __restrict__ match,
                          scalar_t* __restrict__ out) {
    __shared__ scalar_t allsum[512];
    const int Block = 1024;
    __shared__ scalar_t buf[Block * 3];

    for (int i = blockIdx.x; i < b; i += gridDim.x) {
        scalar_t subsum = 0;
        for (int k0 = 0; k0 < n; k0 += blockDim.x) {
            int k = k0 + threadIdx.x;
            scalar_t x1 = 0, y1 = 0, z1 = 0;
            if (k < n) {
                x1 = xyz1[i * n * 3 + k * 3 + 0];
                y1 = xyz1[i * n * 3 + k * 3 + 1];
                z1 = xyz1[i * n * 3 + k * 3 + 2];
            }
            for (int l0 = 0; l0 < m; l0 += Block) {
                int lend = min(m, l0 + Block) - l0;
                for (int l = threadIdx.x; l < lend * 3; l += blockDim.x) {
                    buf[l] = xyz2[i * m * 3 + l0 * 3 + l];
                }
                __syncthreads();
                if (k < n) {
                    for (int l = 0; l < lend; ++l) {
                        scalar_t x2 = buf[l * 3 + 0];
                        scalar_t y2 = buf[l * 3 + 1];
                        scalar_t z2 = buf[l * 3 + 2];
                        scalar_t d  = (x2 - x1) * (x2 - x1) + (y2 - y1) * (y2 - y1) + (z2 - z1) * (z2 - z1);
                        subsum += d * match[i * n * m + (l0 + l) * n + k];
                    }
                }
                __syncthreads();
            }
        }
        allsum[threadIdx.x] = subsum;
        for (int j = 1; j < blockDim.x; j <<= 1) {
            __syncthreads();
            if ((threadIdx.x & j) == 0 && threadIdx.x + j < blockDim.x) {
                allsum[threadIdx.x] += allsum[threadIdx.x + j];
            }
        }
        if (threadIdx.x == 0) out[i] = allsum[0];
        __syncthreads();
    }
}

/* -------------------------------------------------------------------------- */
/* MatchCost forward host interface                                           */
/* -------------------------------------------------------------------------- */

at::Tensor MatchCostForward(const at::Tensor xyz1,
                             const at::Tensor xyz2,
                             const at::Tensor match) {
    const auto b = xyz1.size(0);
    const auto n = xyz1.size(1);
    const auto m = xyz2.size(1);

    TORCH_CHECK_EQ(xyz2.size(0), b);
    TORCH_CHECK_EQ(xyz1.size(2), 3);
    TORCH_CHECK_EQ(xyz2.size(2), 3);
    CHECK_INPUT(xyz1);
    CHECK_INPUT(xyz2);

    auto cost = at::zeros({b}, xyz1.options());

    AT_DISPATCH_FLOATING_TYPES(xyz1.scalar_type(), "MatchCostForward", ([&] {
        matchcost<scalar_t><<<32, 512>>>(
            b, n, m,
            xyz1.data_ptr<scalar_t>(),
            xyz2.data_ptr<scalar_t>(),
            match.data_ptr<scalar_t>(),
            cost.data_ptr<scalar_t>());
    }));
    C10_CUDA_CHECK(hipGetLastError());

    return cost;
}

/********************************
* Backward kernels
*********************************/

template <typename scalar_t>
__global__ void matchcostgrad2(int b, int n, int m,
                               const scalar_t* __restrict__ grad_cost,
                               const scalar_t* __restrict__ xyz1,
                               const scalar_t* __restrict__ xyz2,
                               const scalar_t* __restrict__ match,
                               scalar_t* __restrict__ grad2) {
    __shared__ scalar_t sum_grad[256 * 3];
    for (int i = blockIdx.x; i < b; i += gridDim.x) {
        int kbeg = m * blockIdx.y / gridDim.y;
        int kend = m * (blockIdx.y + 1) / gridDim.y;
        for (int k = kbeg; k < kend; ++k) {
            scalar_t x2 = xyz2[(i * m + k) * 3 + 0];
            scalar_t y2 = xyz2[(i * m + k) * 3 + 1];
            scalar_t z2 = xyz2[(i * m + k) * 3 + 2];
            scalar_t subsumx = 0, subsumy = 0, subsumz = 0;
            for (int j = threadIdx.x; j < n; j += blockDim.x) {
                scalar_t x1 = x2 - xyz1[(i * n + j) * 3 + 0];
                scalar_t y1 = y2 - xyz1[(i * n + j) * 3 + 1];
                scalar_t z1 = z2 - xyz1[(i * n + j) * 3 + 2];
                scalar_t d  = match[i * n * m + k * n + j] * 2;
                subsumx += x1 * d;
                subsumy += y1 * d;
                subsumz += z1 * d;
            }
            sum_grad[threadIdx.x * 3 + 0] = subsumx;
            sum_grad[threadIdx.x * 3 + 1] = subsumy;
            sum_grad[threadIdx.x * 3 + 2] = subsumz;
            for (int j = 1; j < blockDim.x; j <<= 1) {
                __syncthreads();
                int j1 = threadIdx.x;
                int j2 = threadIdx.x + j;
                if ((j1 & j) == 0 && j2 < blockDim.x) {
                    sum_grad[j1 * 3 + 0] += sum_grad[j2 * 3 + 0];
                    sum_grad[j1 * 3 + 1] += sum_grad[j2 * 3 + 1];
                    sum_grad[j1 * 3 + 2] += sum_grad[j2 * 3 + 2];
                }
            }
            if (threadIdx.x == 0) {
                grad2[(i * m + k) * 3 + 0] = sum_grad[0] * grad_cost[i];
                grad2[(i * m + k) * 3 + 1] = sum_grad[1] * grad_cost[i];
                grad2[(i * m + k) * 3 + 2] = sum_grad[2] * grad_cost[i];
            }
            __syncthreads();
        }
    }
}

template <typename scalar_t>
__global__ void matchcostgrad1(int b, int n, int m,
                               const scalar_t* __restrict__ grad_cost,
                               const scalar_t* __restrict__ xyz1,
                               const scalar_t* __restrict__ xyz2,
                               const scalar_t* __restrict__ match,
                               scalar_t* __restrict__ grad1) {
    for (int i = blockIdx.x; i < b; i += gridDim.x) {
        for (int l = threadIdx.x; l < n; l += blockDim.x) {
            scalar_t x1 = xyz1[i * n * 3 + l * 3 + 0];
            scalar_t y1 = xyz1[i * n * 3 + l * 3 + 1];
            scalar_t z1 = xyz1[i * n * 3 + l * 3 + 2];
            scalar_t dx = 0, dy = 0, dz = 0;
            for (int k = 0; k < m; ++k) {
                scalar_t x2 = xyz2[i * m * 3 + k * 3 + 0];
                scalar_t y2 = xyz2[i * m * 3 + k * 3 + 1];
                scalar_t z2 = xyz2[i * m * 3 + k * 3 + 2];
                scalar_t d  = match[i * n * m + k * n + l] * 2;
                dx += (x1 - x2) * d;
                dy += (y1 - y2) * d;
                dz += (z1 - z2) * d;
            }
            grad1[i * n * 3 + l * 3 + 0] = dx * grad_cost[i];
            grad1[i * n * 3 + l * 3 + 1] = dy * grad_cost[i];
            grad1[i * n * 3 + l * 3 + 2] = dz * grad_cost[i];
        }
    }
}

/* -------------------------------------------------------------------------- */
/* MatchCost backward host interface                                          */
/* -------------------------------------------------------------------------- */

std::vector<at::Tensor> MatchCostBackward(const at::Tensor grad_cost,
                                          const at::Tensor xyz1,
                                          const at::Tensor xyz2,
                                          const at::Tensor match) {
    const auto b = xyz1.size(0);
    const auto n = xyz1.size(1);
    const auto m = xyz2.size(1);

    TORCH_CHECK_EQ(xyz2.size(0), b);
    TORCH_CHECK_EQ(xyz1.size(2), 3);
    TORCH_CHECK_EQ(xyz2.size(2), 3);
    CHECK_INPUT(xyz1);
    CHECK_INPUT(xyz2);

    auto grad1 = at::zeros({b, n, 3}, xyz1.options());
    auto grad2 = at::zeros({b, m, 3}, xyz1.options());

    AT_DISPATCH_FLOATING_TYPES(xyz1.scalar_type(), "MatchCostBackward", ([&] {
        matchcostgrad1<scalar_t><<<32, 512>>>(
            b, n, m,
            grad_cost.data_ptr<scalar_t>(),
            xyz1.data_ptr<scalar_t>(),
            xyz2.data_ptr<scalar_t>(),
            match.data_ptr<scalar_t>(),
            grad1.data_ptr<scalar_t>());

        matchcostgrad2<scalar_t><<<dim3(32, 32), 256>>>(
            b, n, m,
            grad_cost.data_ptr<scalar_t>(),
            xyz1.data_ptr<scalar_t>(),
            xyz2.data_ptr<scalar_t>(),
            match.data_ptr<scalar_t>(),
            grad2.data_ptr<scalar_t>());
    }));
    C10_CUDA_CHECK(hipGetLastError());

    return {grad1, grad2};
}

#endif  // _EMD_KERNEL
